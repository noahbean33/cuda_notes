#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1024*1024*1024*4  // Define the size of the vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *A, int *B, int *C, long long  n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int *A, *B, *C;            // Host vectors
    int *d_A, *d_B, *d_C;      // Device vectors
    //long long size = SIZE * sizeof(int);
    const long long size1 = 1024LL * 1024 * 1024 *3;
const long long size = size1 * sizeof(long);
    // CUDA event creation, used for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t err;

    // Allocate and initialize host vectors
    
err=hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed0: %s\n", hipGetErrorString(err));
    }
    err=hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed1: %s\n", hipGetErrorString(err));
    }
    // Allocate device vectors
    err=hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed2: %s\n", hipGetErrorString(err));
    }
A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    for (int i = 0; i < size1; i++) {
        A[i] = i;
        B[i] = i+2;
    }

    // Copy host vectors to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Start recording
    hipEventRecord(start);
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 96;
    int blocksPerGrid = (size1 + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size1);

    // Stop recording
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate and print the execution time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f milliseconds\n", milliseconds);
    for(int i=0;i<10;i++){
        printf("A=%d\tB=%d -------> C=%d  \n",A[i],B[i],C[i]);
    }
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
