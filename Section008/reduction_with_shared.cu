#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reduce_in_place(float* input, int n) {
    __shared__ float shared[1024];  // Shared memory array for this block
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements from global memory to shared memory
        shared[tid] = input[index];

    __syncthreads();  // Synchronize all threads in the block

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();  // Ensure all threads have completed the previous iteration
        //int x = 2 * stride * tid;
        if (index+ stride < n) {
            shared[tid] += shared[tid + stride];
        }
    }
    
    // Write the block's result to global memory
    if (tid == 0) {
        input[blockIdx.x] = shared[0];
    }

}

float cpu_reduce(float* input, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        sum += input[i];
    }
    return sum;
}

int main() {
    int n = 1024*1024;  // Number of elements
    size_t bytes = n * sizeof(float);

    // Host memory allocation
    float* h_input = new float[n];
    float* d_input;

    // Initialize input array
    for (int i = 0; i < n; i++) {
        h_input[i] = static_cast<float>(i + 1);  // Initialize from 1 to n
    }

    // Device memory allocation
    hipMalloc(&d_input, bytes);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Launch the reduction kernel multiple times
    int blockSize = 256;  // Number of threads per block
    int gridSize = (n + blockSize - 1) / blockSize;  // Number of blocks
    // Calculate the sum using the CPU function for verification
    std::cout << "grid size is   " << gridSize << std::endl;
    float total_sum = cpu_reduce(h_input, n);
    std::cout << "Total sum (CPU): " << total_sum << std::endl;
    // Perform iterative reduction until we have one block left
    while (gridSize > 1) {
        reduce_in_place << <gridSize, blockSize >> > (d_input, n);
        hipDeviceSynchronize();  // Ensure kernel execution completes

        // Update n to reflect the reduced number of elements
        n = gridSize;
        gridSize = (n + blockSize - 1) / blockSize;  // Update gridSize for the next iteration
    }

    // Final reduction when gridSize == 1
    reduce_in_place << <1, blockSize >> > (d_input, n);
    hipDeviceSynchronize();  // Ensure final kernel execution completes

    // Copy the final result back to the host (the sum should be in h_input[0])
    hipMemcpy(h_input, d_input, 4*sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Final sum (GPU): " << h_input[0] << std::endl;




    // Free memory
    hipFree(d_input);
    delete[] h_input;

    return 0;
}
