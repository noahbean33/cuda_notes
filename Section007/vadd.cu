#include <hip/hip_runtime.h>
#include <iostream>

const int N = 1024 * 1024 * 32; // Size of the vectors

__global__ void vector_add_1_element(float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    h_c = (float*)malloc(N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Launch kernel
    vector_add_1_element<<<gridSize, blockSize>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
