#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1024*1024*32  // Define the size of the vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *A, int *B, int *C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int *A, *B, *C;            // Host vectors
    int *d_A, *d_B, *d_C;      // Device vectors
    int size = SIZE * sizeof(int);

    // CUDA event creation, used for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate and initialize host vectors
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    for (int i = 0; i < SIZE; i++) {
        A[i] = i;
        B[i] = SIZE - i;
    }

    // Allocate device vectors
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy host vectors to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Start recording
    hipEventRecord(start);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 96;
    int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, SIZE);

    // Stop recording
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate and print the execution time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f milliseconds\n", milliseconds);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
