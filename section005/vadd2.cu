    #include <stdio.h>
    #include <stdlib.h>
    #include <cstdlib>  // for rand()
    #include <hip/hip_runtime.h>

    // Error checking macro
    #define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
    }

    #define gpuKernelCheck() { gpuKernelAssert(__FILE__, __LINE__); }
    inline void gpuKernelAssert(const char *file, int line, bool abort=true) {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s %s %d\n", hipGetErrorString(err), file, line);
            if (abort) exit(err);
        }
    }

    // CUDA Kernel for vector addition
    __global__ void vectorAdd(int *A, int *B, int *C, int n) {
        int ix = (threadIdx.x + blockDim.x * blockIdx.x)*2;
        if(ix<n){
        C[ix] = A[ix] + B[ix];
        C[ix+1] = A[ix+1] + B[ix+1];
        }
    }


    int main() {
        // Check if block size is provided as a command line argument


        // Parse block size from command line
        int threadsPerBlock = 256;//atoi(argv[1]);


        int *A, *B, *C;            // Host vectors
        int *d_A, *d_B, *d_C;      // Device vectors
        long long SIZE = 1024LL * 1024 * 32;
        long size = SIZE * sizeof(int);

        // CUDA event creation, used for timing
        // Allocate device vectors
        cudaCheckError(hipMalloc((void **)&d_A, size));
        cudaCheckError(hipMalloc((void **)&d_B, size));
        cudaCheckError(hipMalloc((void **)&d_C, size));

        // Allocate and initialize host vectors
        A = (int *)malloc(size);
        B = (int *)malloc(size);
        C = (int *)malloc(size);
        for (int i = 0; i < SIZE; i++) {
            A[i] = 10*static_cast<float>(rand()) / static_cast<float>(RAND_MAX);  // Random values between 0 and 1
            B[i] = 20*static_cast<float>(rand()) / static_cast<float>(RAND_MAX);  // Random values between 0 and 1
        }

        // Copy host vectors to device
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        // Launch the Vector Add CUDA Kernel
        int blocksPerGrid = ((SIZE + threadsPerBlock - 1) / threadsPerBlock)/2;
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, SIZE);
        gpuKernelCheck();

        // Copy result back to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        //for (int i=0;i<=128;i++){
        //    printf("\n%d + %d = %d",A[i],B[i],C[i]);
        //}

        for(int i=0;i<=1024LL * 1024 * 32;i++){
            if(C[i]!=A[i]+B[i]){
                printf("\nError in index i %d",i);
                printf("\n%d + %d = %d",A[i],B[i],C[i]);
            }
        }

        // Calculate and print the execution time

        // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(A);
        free(B);
        free(C);

        return 0;
    }

