    #include <stdio.h>
    #include <stdlib.h>
    #include <cstdlib>  // for rand()
    #include <hip/hip_runtime.h>

    // Error checking macro
    #define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
    }

    #define gpuKernelCheck() { gpuKernelAssert(__FILE__, __LINE__); }
    inline void gpuKernelAssert(const char *file, int line, bool abort=true) {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s %s %d\n", hipGetErrorString(err), file, line);
            if (abort) exit(err);
        }
    }

    // CUDA Kernel for vector addition
    __global__ void vectorAdd(int *A, int *B, int *C, int n) {
        int ix = (threadIdx.x + blockDim.x * blockIdx.x);
        if(ix<n){
            C[ix] = A[ix] + B[ix];
        }
    }

    int main() {

        int *A, *B, *C;            // Host vectors
        int *d_A, *d_B, *d_C;      // Device vectors
        long long SIZE = 1024LL * 1024 * 32;
        long size = SIZE * sizeof(int);

        // CUDA event creation, used for timing
        // Allocate device vectors
        cudaCheckError(hipMalloc((void **)&d_A, size));
        cudaCheckError(hipMalloc((void **)&d_B, size));
        cudaCheckError(hipMalloc((void **)&d_C, size));

        // Allocate and initialize host vectors
        A = (int *)malloc(size);
        B = (int *)malloc(size);
        C = (int *)malloc(size);
        for (int i = 0; i < SIZE; i++) {
            A[i] = 10*static_cast<float>(rand()) / static_cast<float>(RAND_MAX);  // Random values between 0 and 1
            B[i] = 20*static_cast<float>(rand()) / static_cast<float>(RAND_MAX);  // Random values between 0 and 1
        }

        // Copy host vectors to device
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = ((SIZE + threadsPerBlock - 1) / threadsPerBlock);
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, SIZE);
        gpuKernelCheck();

        // Copy result back to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        //for (int i=0;i<=128;i++){
        //    printf("\n%d + %d = %d",A[i],B[i],C[i]);
        //}

        for(int i=0;i<=1024LL * 1024 * 32;i++){
            if(C[i]!=A[i]+B[i]){
                printf("\nError in index i %d",i);
                printf("\n%d + %d = %d",A[i],B[i],C[i]);
            }
        }

        // Calculate and print the execution time

        // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(A);
        free(B);
        free(C);

        return 0;
    }

